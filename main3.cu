#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>


#include <time.h>

__global__ void calculations_inv(float *d_array, float *ans,int *n, long *iter)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	float *d_ptr_array = d_array;

	int i;
	//obliczanie sumy
	//ustawienie wskaznika na odpowiedni wiersz tablicy 

	if(id < *iter)
	{

		float add = 0.0;
		float mult = 1.0;

		d_ptr_array = d_ptr_array + id; // przejscie do wiersza "id"
		for(i = 0 ; i < *n; i++)
		{
			add += *d_ptr_array * *d_ptr_array;
			d_ptr_array++;
		}

		//ustawienie wskaznika ponownie na poczatek tablicy 
		d_ptr_array = d_array + id;
		
		mult = cos(*d_ptr_array);
		d_ptr_array++;

		for(i = 1 ; i < *n; i++)
		{
			mult = mult * cos(*d_ptr_array/(i+1));
			d_ptr_array++;
		}

		ans[id] = 1 / 40 * add + 1 - mult;
	}
}

int main(int argc, char const *argv[])
{
	if(argc == 5)
	{

		clock_t start, end;
		double used_time;

		start = clock();

		int i; 									 // iterator
		int j;								     // iterator

		//zmienne z lini argumentów wywołania programu
		const int n 					= atoi(argv[1]); // wymiar zadania
		const int I 					= atoi(argv[2]); // liczba iteracji - > obliczenia przeprowadzane sa na wartosci I^n
		const double iter 				= pow(I, n);
		const int Blocks 				= atoi(argv[3]); // liczba bloków GPU
		const int Threads 				= atoi(argv[4]); // liczba watków dla jednego bloku GPU

		//const float x_min 				= -20.0; // minimalna warotsc dziedziny zadania
		const float x_max 				= 20.0; // maksymalna wartosc dziedziny zadania
		const float rand_max 			= RAND_MAX / 40.0; //ograniczenie przedzialu losowania zmiennych

		float *h_random_array = (float* ) malloc(sizeof(float) * n * iter); //do operacji na danych stosowana jest tablica jednowymiarowa ze wzgledu na alokacje pamieci w GPU
		float *h_ptr_iterator = h_random_array;

		float *ans = (float* ) malloc(sizeof(float) * iter);

		//losowanie wartosci i umieszczenie ich w tablicy
		for(i = 0 ; i < iter; i++)
		{
			for(j = 0 ; j < n ; j++)
			{
				*h_ptr_iterator = rand() / rand_max - x_max;
				h_ptr_iterator += 1;
			}
		}

		float *d_random_array; //tablica zmiennych wylosowanych w pamieci GPU
		float *d_ans;  //tablica wynikow
		int *d_n; // wymiar 
		long *d_iter; //ilosc iteratcji

		hipMalloc((void **)&d_random_array, sizeof(float) * n * iter);
		hipMalloc((void **)&d_ans, sizeof(float) * iter);
		hipMalloc((void **)&d_n, sizeof(int));
		hipMalloc((void **)&d_iter, sizeof(long));

		hipMemcpy(d_random_array, h_random_array, sizeof(float) * n * iter, hipMemcpyHostToDevice);
		hipMemcpy(d_ans, ans, sizeof(float) * iter, hipMemcpyHostToDevice);
		hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_iter, &iter, sizeof(long) , hipMemcpyHostToDevice);

		printf("Liczba blocków : n [%d] Liczba iteracji [%f] , bloki [%d] watki : [%d]", n , iter, Blocks , Threads);

		calculations_inv<<<Blocks, Threads>>>(d_random_array, d_ans, d_n, d_iter);

		hipMemcpy(ans, d_ans, sizeof(float) * iter , hipMemcpyDeviceToHost);

		//szukanie minimum

		float y_min  = ans[0];
		for(i = 0 ; i < iter; i++)
		{
			if(ans[i] < y_min) y_min = ans[i];
		}

		end = clock();
		used_time = ((double) (end - start) / CLOCKS_PER_SEC);

		printf("szukane minimum : %f - czas : %f  \n " , y_min, used_time);

		hipFree(d_random_array);
		hipFree(d_ans);
		hipFree(d_n);
		hipFree(d_iter);

		free(h_random_array);
		free(ans);
	}
	else
	{
		printf("Invalid program parameters plese type /main2 N I Blocks Threads where \n");
		printf("N - is problem dimension\n");
		printf("I - is number of iteratios\n");
		printf("Blocks - is number of used GPU blocks...max is %d\n", 0);
		printf("Threads- is number of used GPU threads per one block  ... max is %d \n", 0);

		return 0;
	}	

	return 0;
}